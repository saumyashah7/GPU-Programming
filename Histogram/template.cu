#include "hip/hip_runtime.h"
#include <wb.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""

#define NUM_BINS 4096
#define BLOCK_SIZE 512 

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void histogram(unsigned int *input, unsigned int *bins,
	unsigned int num_elements,
	unsigned int num_bins) 
{
	//@@ Write the kernel that computes the histogram
	//@@ Make sure to use the privitization technique
	//(hint: since NUM_BINS=4096 is larger than maximum allowed number of threads per block, 
	//be aware that threads would need to initialize more than one shared memory bin 
	//and update more than one global memory bin)
	__shared__ unsigned int histo[NUM_BINS];
	

    for(int i=0;i<NUM_BINS/BLOCK_SIZE;i++)
    {
		histo[threadIdx.x + blockDim.x*i] = 0;
    } 
    __syncthreads();
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < num_elements) 
	{
        atomicAdd(&(histo[input[i]]), 1);
	}
    
    __syncthreads();
    for(int j=0;j<NUM_BINS/BLOCK_SIZE;j++)
    {
        atomicAdd(&(bins[threadIdx.x + blockDim.x*j]), histo[threadIdx.x + blockDim.x*j]);
    }
}

__global__ void saturate(unsigned int *bins, unsigned int num_bins) 
{
	//@@ Write the kernel that applies saturtion to counters (i.e., if the bin value is more than 127, make it equal to 127)
	for(int i=0;i<NUM_BINS/BLOCK_SIZE;i++)
    {
        if(bins[threadIdx.x + blockDim.x*i] > 127)
        {
            bins[threadIdx.x + blockDim.x*i] = 127;
        } 
    }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating device memory");
  //@@ Allocate device memory here
  (hipMalloc((void**)&deviceInput, inputLength * sizeof(float)));
  (hipMalloc((void**)&deviceBins, NUM_BINS * sizeof(float)));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating device memory");

  wbTime_start(GPU, "Copying input host memory to device");
  //@@ Copy input host memory to device
  (hipMemcpy(deviceInput, hostInput, inputLength * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input host memory to device");
	
  wbTime_start(GPU, "Clearing the bins on device");
  //@@ zero out the deviceBins using hipMemset() 
  hipMemset(deviceBins, 0, inputLength * sizeof(float));
  wbTime_stop(GPU, "Clearing the bins on device");

  //@@ Initialize the grid and block dimensions here
  int numBlocks = ceil((float)inputLength / (BLOCK_SIZE ));
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  wbLog(TRACE, "The number of blocks is ", numBlocks);

  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Invoke kernels: first call histogram kernel and then call saturate kernel
  // histogram(unsigned int *input, unsigned int *bins,unsigned int num_elements,unsigned int num_bins)
  histogram<<<dimGrid, dimBlock>>> (deviceInput,deviceBins,inputLength,NUM_BINS);
  //saturate(unsigned int *bins, unsigned int num_bins)
  saturate<<<(1,1,1),dimBlock>>>(deviceBins,NUM_BINS);

  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy output device memory to host
  (hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Free the device memory here
  hipFree(deviceInput);
  hipFree(deviceBins);
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}