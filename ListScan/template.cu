#include "hip/hip_runtime.h"
#include <wb.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 512 

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, float *aux, int len) {
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array 
	__shared__ float XY[2 * BLOCK_SIZE];

	//loading data from global memory to shared memory
	int i = 2 * blockIdx.x*blockDim.x + threadIdx.x;
	if (i<len)
		XY[threadIdx.x] = input[i];
	else
		XY[threadIdx.x] = 0;


	if (i + blockDim.x<len)
		XY[threadIdx.x + blockDim.x] = input[i + blockDim.x];
	else
		XY[threadIdx.x + blockDim.x] = 0;
		
	__syncthreads();
	
	//Reduction Phase
	for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2)
	{
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index < 2 * BLOCK_SIZE)
			XY[index] += XY[index - stride];
		__syncthreads();
	}

	//Post Reduction Phase
	for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
		int index = (threadIdx.x + 1)*stride * 2 - 1;
		if (index + stride < 2 * BLOCK_SIZE) 
			XY[index + stride] += XY[index];
		
		__syncthreads();
	}
	
	//writing data to output
	if (i < len) 
		output[i] = XY[threadIdx.x];
	if (i + blockDim.x < len) 
		output[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
	if(aux!=NULL && threadIdx.x==0)
		aux[blockIdx.x] = XY[2 * blockDim.x - 1];

     
}

__global__ void addScannedBlockSums(float *input, float *aux, int len) {
	//@@ Modify the body of this kernel to add scanned block sums to 
	//@@ all values of the scanned blocks
	int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

	if (blockIdx.x > 0) {

		if (i < len)
			input[i] += aux[blockIdx.x - 1];

		if (i + blockDim.x < len)
			input[i + blockDim.x] += aux[blockIdx.x - 1];
	}

}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output 1D list
  float *deviceInput;
  float *deviceOutput;
  float *deviceAuxArray, *deviceAuxScannedArray;
  int numElements; // number of elements in the input/output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating device memory.");
  //@@ Allocate device memory
  //you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceAuxScannedArray, 2 * BLOCK_SIZE  * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceAuxArray, 2 * BLOCK_SIZE * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbTime_stop(GPU, "Allocating device memory.");

  wbTime_start(GPU, "Clearing output device memory.");
  //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output device memory.");

  wbTime_start(GPU, "Copying input host memory to device.");
  //@@ Copy input host memory to device	
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input host memory to device.");

  //@@ Initialize the grid and block dimensions here

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //@@ You need to launch scan kernel twice: 1) for generating scanned blocks 
  //@@ (hint: pass deviceAuxArray to the aux parameter)
  //@@ and 2) for generating scanned aux array that has the scanned block sums. 
  //@@ (hint: pass NULL to the aux parameter)
  //@@ Then you should call addScannedBlockSums kernel.
  int numBlocks = ceil(numElements / float(BLOCK_SIZE * 2));

  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  scan<<<dimGrid, dimBlock >> >(deviceInput, deviceOutput, deviceAuxArray, numElements);
  hipDeviceSynchronize();

  dim3 singleGrid(1, 1, 1);
  scan<<<singleGrid, dimBlock >> >(deviceAuxArray, deviceAuxScannedArray, NULL, BLOCK_SIZE * 2);
  hipDeviceSynchronize();

  addScannedBlockSums <<<dimGrid, dimBlock >> >(deviceOutput, deviceAuxScannedArray, numElements);
  hipDeviceSynchronize();

  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy results from device to host	
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Deallocate device memory
  hipFree(deviceInput);
  hipFree(deviceAuxScannedArray);
  hipFree(deviceAuxArray);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
